#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <hip/hip_runtime.h>

#include "macros.h"
#include "cuda_utils.h"

__global__ void grid_interp_cuda_kernel(
    const torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits> vol,
    const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> points,
    torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> output,
    int channels, int3 nGrids, size_t size) {

    const int tx = blockIdx.x * blockDim.x + threadIdx.x;
    const int ty = blockIdx.y * blockDim.y + threadIdx.y;
    const int tz = blockIdx.z * blockDim.z + threadIdx.z;
    const int index = (tz * blockDim.y * gridDim.y + ty) * blockDim.x * gridDim.x + tx;
    if (index >= size) {
        return;
    }

    const float x = points[index][0];
    const float y = points[index][1];
    const float z = points[index][2];

    const int ix = (int)x;
    const int iy = (int)y;
    const int iz = (int)z;
    const float fx = x - ix;
    const float fy = y - iy;
    const float fz = z - iz;

    for (int c = 0; c < channels; c++) {
        const int x0 = max(0, min(ix, nGrids.x - 1));
        const int x1 = max(0, min(ix + 1, nGrids.x - 1));
        const int y0 = max(0, min(iy, nGrids.y - 1));
        const int y1 = max(0, min(iy + 1, nGrids.y - 1));
        const int z0 = max(0, min(iz, nGrids.z - 1));
        const int z1 = max(0, min(iz + 1, nGrids.z - 1));

        const float v00 = (1.0 - fx) * vol[c][z0][y0][x0] + fx * vol[c][z0][y0][x1];
        const float v01 = (1.0 - fx) * vol[c][z0][y1][x0] + fx * vol[c][z0][y1][x1];
        const float v10 = (1.0 - fx) * vol[c][z1][y0][x0] + fx * vol[c][z1][y0][x1];
        const float v11 = (1.0 - fx) * vol[c][z1][y1][x0] + fx * vol[c][z1][y1][x1];

        const float v0 = (1.0 - fy) * v00 + fy * v01;
        const float v1 = (1.0 - fy) * v10 + fy * v11;

        output[index][c] = (1.0 - fz) * v0 + fz * v1;
    }
}

torch::Tensor grid_interp_cuda(torch::Tensor vol, torch::Tensor points) {
    // Check input tensors
    CHECK_CUDA(vol);
    CHECK_CONTIGUOUS(vol);
    CHECK_IS_FLOAT(vol);
    CHECK_N_DIM(vol, 4);

    CHECK_CUDA(points);
    CHECK_CONTIGUOUS(points);
    CHECK_IS_FLOAT(vol);
    CHECK_N_DIM(points, 2);

    // Size parameters
    const int Nx = vol.size(3);
    const int Ny = vol.size(2);
    const int Nz = vol.size(1);
    const int C = vol.size(0);
    const int Np = points.size(0);
    const int deviceId = vol.device().index();

    torch::Tensor output = torch::zeros({Np, C},
        torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, deviceId));

    auto volAscr = vol.packed_accessor32<float, 4, torch::RestrictPtrTraits>();
    auto ptsAscr = points.packed_accessor32<float, 2, torch::RestrictPtrTraits>();
    auto outAscr = output.packed_accessor32<float, 2, torch::RestrictPtrTraits>();

    const uint32_t MAX_THREADS_AXIS = 128;
    const uint32_t MAX_THREADS_AXIS2 = MAX_THREADS_AXIS * MAX_THREADS_AXIS;
    const uint32_t blockx = MAX_THREADS_AXIS;
    const uint32_t blocky = MAX_THREADS_AXIS;
    const uint32_t blockz = (Np + MAX_THREADS_AXIS2 - 1) / MAX_THREADS_AXIS2;

    const uint32_t BLOCK_SIZE = 8;
    const uint32_t gridx = (blockx + BLOCK_SIZE - 1) / BLOCK_SIZE;
    const uint32_t gridy = (blocky + BLOCK_SIZE - 1) / BLOCK_SIZE;
    const uint32_t gridz = (blockz + BLOCK_SIZE - 1) / BLOCK_SIZE;
    const int3 nGrids = make_int3(Nx, Ny, Nz);

    const dim3 blocks = { gridx, gridy, gridz };
    const dim3 threads = { BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE };
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    grid_interp_cuda_kernel<<<blocks, threads, 0, stream>>>(volAscr, ptsAscr, outAscr, C, nGrids, Np);

    CUDA_CHECK_ERRORS();

    return output;
}
