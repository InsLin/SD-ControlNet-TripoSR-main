#include "hip/hip_runtime.h"
#include "pscan.h"

#include <iostream>

#include <hip/hip_runtime.h>

#include "cuda_utils.h"

static const int THREADS_PER_BLOCK = 128;
static const int ELEMENTS_PER_BLOCK = THREADS_PER_BLOCK * 2;

__host__ int nextPowerOfTwo(int x) {
    int power = 1;
    while (power < x) {
        power *= 2;
    }
    return power;
}

__global__ void prescan_small_kernel(int *input, int *output, int n, int pow2) {
    extern __shared__ int buffer[];
    const int threadID = threadIdx.x;

    if (threadID < n) {
        buffer[2 * threadID] = input[2 * threadID];
        buffer[2 * threadID + 1] = input[2 * threadID + 1];
    } else {
        buffer[2 * threadID] = 0.0;
        buffer[2 * threadID + 1] = 0.0;
    }

    int offset = 1;
    for (int d = pow2 >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (threadID < d) {
            const int ai = offset * (2 * threadID + 1) - 1;
            const int bi = offset * (2 * threadID + 2) - 1;
            buffer[bi] += buffer[ai];
        }
        offset *= 2;
    }

    if (threadID == 0) { buffer[pow2 - 1] = 0; }

    for (int d = 1; d < pow2; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (threadID < d) {
            const int ai = offset * (2 * threadID + 1) - 1;
            const int bi = offset * (2 * threadID + 2) - 1;
            const int t = buffer[ai];
            buffer[ai] = buffer[bi];
            buffer[bi] += t;           
        }
    }
    __syncthreads();

    if (threadID < n) {
        output[2 * threadID] = buffer[2 * threadID];
        output[2 * threadID + 1] = buffer[2 * threadID + 1];
    }
}

__global__ void prescan_large_kernel(int *input, int *output, int n, int *sums) {
    const int blockID = blockIdx.x;
    const int threadID = threadIdx.x;
    const int blockOffset = blockID * n;

    extern __shared__ int buffer[];
    buffer[2 * threadID] = input[blockOffset + (2 * threadID)];
    buffer[2 * threadID + 1] = input[blockOffset + (2 * threadID + 1)];

    int offset = 1;
    for (int d = n >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (threadID < d) {
            const int ai = offset * (2 * threadID + 1) - 1;
            const int bi = offset * (2 * threadID + 2) - 1;
            buffer[bi] += buffer[ai];
        }
        offset *= 2;
    }
    __syncthreads();

    if (threadID == 0) {
        sums[blockID] = buffer[n - 1];
        buffer[n - 1] = 0;
    }

    for (int d = 1; d < n; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (threadID < d) {
            const int ai = offset * (2 * threadID + 1) - 1;
            const int bi = offset * (2 * threadID + 2) - 1;
            const int t = buffer[ai];
            buffer[ai] = buffer[bi];
            buffer[bi] += t;           
        }
    }
    __syncthreads();

    output[blockOffset + (2 * threadID)] = buffer[2 * threadID];
    output[blockOffset + (2 * threadID + 1)] = buffer[2 * threadID + 1];
}

__global__ void add(int *output, int length, int *n) {
    const int blockID = blockIdx.x;
    const int threadID = threadIdx.x;
    const int blockOffset = blockID * length;
    output[blockOffset + threadID] += n[blockID];
}

__global__ void add(int *output, int length, int *n1, int *n2) {
    const int blockID = blockIdx.x;
    const int threadID = threadIdx.x;
    const int blockOffset = blockID * length;
    output[blockOffset + threadID] += n1[blockID] + n2[blockID];
}

void prescan_small(int *d_in, int *d_out, int n, int dev_id = 0, hipStream_t stream = 0) {
    const int pow2 = nextPowerOfTwo(n);
    hipSetDevice(dev_id);
    prescan_small_kernel<<<1, (n + 1) / 2, 2 * pow2 * sizeof(int), stream>>>(d_in, d_out, n, pow2);

    CUDA_CHECK_ERRORS();
}

void prescan_large(int *d_in, int *d_out, int n, int dev_id = 0, hipStream_t stream = 0) {
    const int blocks = (n + ELEMENTS_PER_BLOCK - 1) / ELEMENTS_PER_BLOCK;
    const int sharedSize = ELEMENTS_PER_BLOCK * sizeof(int);

    hipSetDevice(dev_id);
    int *d_sums, *d_incr;
    hipMalloc((void **)&d_sums, blocks * sizeof(int));
    hipMalloc((void **)&d_incr, blocks * sizeof(int));

    prescan_large_kernel<<<blocks, THREADS_PER_BLOCK, 2 * sharedSize, stream>>>(
        d_in, d_out, ELEMENTS_PER_BLOCK, d_sums);

    const int sumThreadsNeeded = (blocks + 1) / 2;
    if (sumThreadsNeeded > THREADS_PER_BLOCK) {
        prescan_large(d_sums, d_incr, blocks, dev_id, stream);
    } else {
        prescan_small(d_sums, d_incr, blocks, dev_id, stream);
    }

    add<<<blocks, ELEMENTS_PER_BLOCK, 0, stream>>>(d_out, ELEMENTS_PER_BLOCK, d_incr);

    hipSetDevice(dev_id);
    hipFree(d_sums);
    hipFree(d_incr);

    CUDA_CHECK_ERRORS();
}

void prescan(int *d_in, int *d_out, int size, int dev_id, hipStream_t stream) {
    const size_t residue = size % ELEMENTS_PER_BLOCK;
    if (size < ELEMENTS_PER_BLOCK) {
        prescan_small(d_in, d_out, size, dev_id, stream);
    } else if (residue == 0) {
        prescan_large(d_in, d_out, size, dev_id, stream);
    } else {
        const size_t tail = size - residue;
        prescan_large(d_in, d_out, tail, dev_id, stream);
        prescan_small(&d_in[tail], &d_out[tail], residue, dev_id, stream);
        add<<<1, residue, 0, stream>>>(&d_out[tail], residue, &d_in[tail - 1], &d_out[tail - 1]);
    }

    CUDA_CHECK_ERRORS();
}
